#include "hip/hip_runtime.h"
#undef _SCRT_STARTUP_MAIN
#include <iostream>	

#include "opencv2/opencv.hpp"

#include "glm/glm.hpp"
#include "glm/ext.hpp"
#include <chrono>
#include <fstream>
#include <strstream>
#include <math.h>
#include <Windows.h>
#include "lua.hpp"
#include <vector>
#include <algorithm>
#include <strstream>
#include <filesystem>
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "cudamathOld.cuh"
#include <thrust/device_vector.h>
#include "OpenImageDenoise/oidn.hpp"
#define RESOLUTION 512
#define ALLOC_MEM_TRIS_NUM 800
__host__ __device__ struct Triangle
{
public:
	float points[13] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
	float3 col = { 0, 1, 0 };
	float3 normal = { 0, 0, 0 };
};
void calculateBounding(Triangle& triangle)
{
	float* points = triangle.points;
	triangle.points[9] = min(points[0], min(points[3], points[6]));
	triangle.points[10] = max(points[0], max(points[3], points[6]));
	triangle.points[11] = min(points[1], min(points[4], points[7]));
	triangle.points[12] = max(points[1], max(points[4], points[7]));
}
void calculateNormal(Triangle& triangle)
{
	float3 normal = normalize(cross(sub({ triangle.points[6], triangle.points[7], triangle.points[8] }, { triangle.points[0], triangle.points[1], triangle.points[2] }), sub({ triangle.points[3], triangle.points[4], triangle.points[5] }, { triangle.points[0], triangle.points[1], triangle.points[2] })));
	triangle.normal = normal;
}
__host__ __device__ struct Mesh
{
public:
	glm::vec3 position;
	std::vector<Triangle> triangles;
};
__host__ __device__ bool rayTriangleIntersect(float3 v0, float3 v1, float3 v2, float& t, float3 rayPos, float3 rayVec)
{
	float3 dir = rayVec;
	float3 orig = rayPos;
	float3 v0v1 = sub(v1, v0);
	float3 v0v2 = sub(v2, v0);
	float3 N = cross(v0v1, v0v2);
	float area2 = length(N);
	float NdotRayDirection = dot(N, dir);
	if (fabs(NdotRayDirection) < 0.000001f)
		return false;
	float d = -dot(N, v0);
	t = -(dot(N, orig) + d) / NdotRayDirection;
	if (t < 0) return false;
	float3 P = add(orig, mult({ t, t, t }, dir));
	float3 C;
	float3 edge0 = sub(v1, v0);
	float3 vp0 = sub(P, v0);
	C = cross(edge0, vp0);
	if (dot(N, C) < 0) return false;
	float3 edge1 = sub(v2, v1);
	float3 vp1 = sub(P, v1);
	C = cross(edge1, vp1);
	if (dot(N, C) < 0)  return false;
	float3 edge2 = sub(v0, v2);
	float3 vp2 = sub(P, v2);
	C = cross(edge2, vp2);
	if (dot(N, C) < 0) return false;
	return true;
}
__global__ void _draw_pix(int y, float camX, float camY, float camZ, float sunX, float sunY, float sunZ, float playerX, float playerY, float playerZ, float degreesXZ, float degreesYZ, hiprandState* rand_state, Triangle* tris, int depth, int* r, int* g, int* b)
{
	float i = threadIdx.x;
	float j = blockIdx.x;
	float3 rayVec = normalize({ -(i / RESOLUTION - 0.5f), -((j + y * RESOLUTION * 0.25f) / RESOLUTION - 0.5f), 1.0f });
	float3 rayPos{ camX, camY, camZ };
	int3 fragColor{ 255, 100, 100 };
	float SINF = sinf(degreesXZ);
	float COSF = cosf(degreesXZ);
	float output2[1][3] = { { 0, 0, 0 } };
	float input1[1][3];
	float input2[3][3];
	input2[0][0] = 1;
	input2[0][1] = 0;
	input2[0][2] = 0;
	input2[1][0] = 0;
	input2[2][0] = 0;
	input2[1][1] = COSF;
	input2[1][2] = -SINF;
	input2[2][1] = SINF;
	input2[2][2] = COSF;
	input1[0][0] = rayVec.x;
	input1[0][1] = rayVec.y;
	input1[0][2] = rayVec.z;
	for (int _ = 0;_ < 1;_++)
		for (int Y = 0;Y < 3;Y++)
			for (int k = 0;k < 3;k++)
			{
				output2[_][Y] += input1[_][k] * input2[k][Y];
			}
	rayVec = { (float)output2[0][0], (float)output2[0][1], (float)output2[0][2] };
	SINF = sinf(degreesYZ);
	COSF = cosf(degreesYZ);
	float output22[1][3] = { { 0, 0, 0 } };
	input2[0][0] = COSF;
	input2[0][1] = 0;
	input2[0][2] = SINF;
	input2[1][0] = 0;
	input2[2][0] = -SINF;
	input2[1][1] = 1;
	input2[1][2] = 0;
	input2[2][1] = 0;
	input2[2][2] = COSF;
	input1[0][0] = rayVec.x;
	input1[0][1] = rayVec.y;
	input1[0][2] = rayVec.z;
	for (int _ = 0;_ < 1;_++)
		for (int Y = 0;Y < 3;Y++)
			for (int k = 0;k < 3;k++)
			{
				output22[_][Y] += input1[_][k] * input2[k][Y];
			}
	rayVec = { (float)output22[0][0], (float)output22[0][1], (float)output22[0][2] };
	rayVec = normalize(rayVec);
	Triangle closest;
	float closeT = 1000.0f;
	for (int i = 0;i < ALLOC_MEM_TRIS_NUM;i++)
	{
		float3 p1{ tris[i].points[0], tris[i].points[1], tris[i].points[2] };
		float3 p2{ tris[i].points[3], tris[i].points[4], tris[i].points[5] };
		float3 p3{ tris[i].points[6], tris[i].points[7], tris[i].points[8] };
		float t;
		if (rayTriangleIntersect(p1, p2, p3, t, rayPos, rayVec))
		{
			if (t < closeT)
			{
				closeT = t;
				closest = tris[i];
			}
		}
	}
	bool playerHit = false;
	float3 oc = sub(rayPos, { playerX, playerY, playerZ });
	float a = dot(rayVec, rayVec);
	float B = 2.0 * dot(oc, rayVec);
	float c = dot(oc, oc) - 0.2f * 0.2f;
	float discriminant = B * B - 4 * a * c;
	if (discriminant >= 0.0) {
		float numerator = -B - sqrtf(discriminant);
		if (numerator > 0.0) {
			float dist = numerator / (2.0 * a);
			if (dist < closeT)
			{
				playerHit = true;
				float3 intersect = add(rayPos, mult({ dist, dist, dist }, rayVec));
				float3 normal = normalize(sub({ playerX, playerY, playerZ }, intersect));
				float lums = max(dot(normal, normalize(sub(intersect, { sunX, sunY, sunZ }))), 0.0f) * 255;
				fragColor = { (int)(lums * 0), (int)(lums * 1), (int)(lums * 0) };
			}
		}
	}
	if (closeT < 1000.0f && !playerHit)
	{
		float3 intersect = add(rayPos, mult({ closeT, closeT, closeT }, rayVec));
		float totalR = 0, totalG = 0, totalB = 0;
		for (int n = 0;n < depth;n++)
		{
			rayVec = normalize(sub(normalize(sub({ sunX, sunY, sunZ }, intersect)), { hiprand_uniform(rand_state + n + threadIdx.x) * 0.5f - 0.25f, hiprand_uniform(rand_state + n + 1 + blockIdx.x) * 0.5f - 0.25f, hiprand_uniform(rand_state + 2 + n + threadIdx.x + blockIdx.x) * 0.5f - 0.25f }));
			rayPos = add(intersect, mult(rayVec, { 0.005f, 0.005f, 0.005f }));
			closeT = 1000.0f;
			Triangle closest2;
			for (int i = 0;i < ALLOC_MEM_TRIS_NUM;i++)
			{
				float3 p1{ tris[i].points[0], tris[i].points[1], tris[i].points[2] };
				float3 p2{ tris[i].points[3], tris[i].points[4], tris[i].points[5] };
				float3 p3{ tris[i].points[6], tris[i].points[7], tris[i].points[8] };
				float t;
				if (rayTriangleIntersect(p1, p2, p3, t, rayPos, rayVec))
				{
					if (t < closeT)
					{
						closeT = t;
						closest2 = tris[i];
					}
				}
			}
			float3 oc = sub(rayPos, { playerX, playerY, playerZ });
			float a = dot(rayVec, rayVec);
			float B = 2.0 * dot(oc, rayVec);
			float c = dot(oc, oc) - 0.2f * 0.2f;
			float discriminant = B * B - 4 * a * c;
			if (discriminant >= 0.0) {
				float numerator = -B - sqrtf(discriminant);
				if (numerator > 0.0) {
					float dist = numerator / (2.0 * a);
					if (dist < closeT)
					{
						closeT = dist;
					}
				}
			}
			float lums = 0;
			if (closeT < 1000.0f)
			{
				lums = max(dot(closest.normal, normalize(sub(intersect, { sunX, sunY, sunZ }))), 0.0f) * 40;
			}
			else
			{
				lums = max(dot(closest.normal, normalize(sub(intersect, { sunX, sunY, sunZ }))), 0.0f) * 255;
			}
			totalR += (lums * closest.col.x);
			totalG += (lums * closest.col.y);
			totalB += (lums * closest.col.z);
		}
		totalR /= depth;
		totalG /= depth;
		totalB /= depth;
		fragColor = { (int)totalR, (int)totalG, (int)totalB };
	}

	r[(int)i + (int)j * RESOLUTION] = fragColor.x;
	g[(int)i + (int)j * RESOLUTION] = fragColor.y;
	b[(int)i + (int)j * RESOLUTION] = fragColor.z;
}
class Wrapper
{
public:
	int* r;
	int* g;
	int* b;
};
bool loadFromObjectFile(std::string sFilename, std::vector<Triangle>& anyData)
{
	std::ifstream f(sFilename);
	if (!f.is_open())
		return false;

	// Local cache of verts
	std::vector<glm::vec3> verts;
	int i = -1;
	while (!f.eof())
	{
		char line[128];
		f.getline(line, 128);

		std::strstream s;
		s << line;

		char junk;

		if (line[0] == 'v')
		{
			glm::vec3 v;
			s >> junk >> v[0] >> v[1] >> v[2];
			verts.push_back(v);
		}

		if (line[0] == 'f')
		{
			i++;
			int f[3];
			s >> junk >> f[0] >> f[1] >> f[2];
			if (i < ALLOC_MEM_TRIS_NUM)
			{
				anyData[i].points[0] = verts[f[0] - 1][0];
				anyData[i].points[1] = verts[f[0] - 1][1];
				anyData[i].points[2] = verts[f[0] - 1][2] + 10.0f;
				anyData[i].points[3] = verts[f[1] - 1][0];
				anyData[i].points[4] = verts[f[1] - 1][1];
				anyData[i].points[5] = verts[f[1] - 1][2] + 10.0f;
				anyData[i].points[6] = verts[f[2] - 1][0];
				anyData[i].points[7] = verts[f[2] - 1][1];
				anyData[i].points[8] = verts[f[2] - 1][2] + 10.0f;
				anyData[i].col = { (float)(rand() % 255) / 255, (float)(rand() % 255) / 255, (float)(rand() % 255) / 255 };
				calculateBounding(anyData[i]);
				calculateNormal(anyData[i]);
			}
		}
	}

	return true;
}
Wrapper helper(int y, float camX, float camY, float camZ, float sunX, float sunY, float sunZ, float playerX, float playerY, float playerZ, float degreesXZ, float degreesYZ, Triangle* tris, hiprandState* state, int depth)
{
	Triangle* dev_tris = nullptr;
	hipMalloc(&dev_tris, sizeof(Triangle) * ALLOC_MEM_TRIS_NUM);
	hipMemcpy(dev_tris, tris, sizeof(Triangle) * ALLOC_MEM_TRIS_NUM, hipMemcpyHostToDevice);
	int* dev_outputR = nullptr;
	int outputR[(RESOLUTION * (RESOLUTION / 4))] = { 255 };
	hipMalloc(&dev_outputR, (RESOLUTION * (RESOLUTION / 4)) * sizeof(int));
	hipMemcpy(dev_outputR, outputR, (RESOLUTION * (RESOLUTION / 4)) * sizeof(int), hipMemcpyHostToDevice);
	int* dev_outputG = nullptr;
	int outputG[(RESOLUTION * (RESOLUTION / 4))] = { 255 };
	hipMalloc(&dev_outputG, (RESOLUTION * (RESOLUTION / 4)) * sizeof(int));
	hipMemcpy(dev_outputG, outputG, (RESOLUTION * (RESOLUTION / 4)) * sizeof(int), hipMemcpyHostToDevice);
	int* dev_outputB = nullptr;
	int outputB[(RESOLUTION * (RESOLUTION / 4))] = { 255 };
	hipMalloc(&dev_outputB, (RESOLUTION * (RESOLUTION / 4)) * sizeof(int));
	hipMemcpy(dev_outputB, outputB, (RESOLUTION * (RESOLUTION / 4)) * sizeof(int), hipMemcpyHostToDevice);
	hiprandState* dev_state = nullptr;
	hipMalloc(&dev_state, sizeof(hiprandState));
	hipMemcpy(dev_state, state, sizeof(hiprandState), hipMemcpyHostToDevice);
	_draw_pix << <RESOLUTION / 4, RESOLUTION >> > (y, camX, camY, camZ, sunX, sunY, sunZ, playerX, playerY, playerZ, degreesXZ, degreesYZ, dev_state, dev_tris, depth, dev_outputR, dev_outputG, dev_outputB);
	hipDeviceSynchronize();
	hipMemcpy(outputR, dev_outputR, RESOLUTION * (RESOLUTION / 4) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(outputG, dev_outputG, RESOLUTION * (RESOLUTION / 4) * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(outputB, dev_outputB, RESOLUTION * (RESOLUTION / 4) * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_tris);
	hipFree(dev_outputR);
	hipFree(dev_outputG);
	hipFree(dev_outputB);
	Wrapper wrapper;
	wrapper.r = outputR;
	wrapper.g = outputG;
	wrapper.b = outputB;
	return wrapper;
}
float camPos[3] = { 0, 0, 0 };
float camRot[3] = { -0.0, 0, 0 };
float sunPos[3] = { 0.0f, 10.0f, 10.0f };
float mouseDiff[2] = { 0, 0 };
void mouseCallback(int event, int x, int y, int flags, void* userData)
{
	mouseDiff[0] = (float)x / RESOLUTION * glm::two_pi<float>() * 2;
	mouseDiff[1] = (float)y / RESOLUTION * glm::two_pi<float>() * 2;
}
void main()
{
	srand(time(NULL));
	Mesh mesh;
	mesh.triangles = std::vector<Triangle>(ALLOC_MEM_TRIS_NUM);
	loadFromObjectFile("C:/Users/arthu/ObjFiles/helmet.obj", mesh.triangles);
	cv::Mat canvas;

	glm::mat4 projection = glm::perspectiveFov(glm::half_pi<float>() / 2.0f, 2.0f, 2.0f, 0.01f, 100.0f);

	float frameCount = 0;
	glm::mat4 identity = glm::identity<glm::mat4>();

	float playerPos[3] = { 0, 2.0f, 0 };
	float playerVec[3] = { 0, 0, 0 };
	float playerRotY = 0.0f;
	hiprandState* state;
	float cameraDist = 4.0f;
	float depth_UNDERCOVER = 1.0f;
	int depth = 1.0f;
	oidn::DeviceRef device = oidn::newDevice();
	device.commit();
	oidn::FilterRef filter = device.newFilter("RT"); // generic ray tracing filter
	while (true)
	{
		frameCount++;
		canvas = cv::Mat::zeros(cv::Size(RESOLUTION, RESOLUTION), CV_8UC3);
		std::vector<Triangle> oldTris = mesh.triangles;
		float s = clock();
		glm::quat rotQuat = glm::angleAxis(camRot[0], glm::vec3{ 1.0f, 0.0f, 0.0f });
		glm::mat4 rotMatX = glm::mat4_cast(rotQuat);
		rotQuat = glm::angleAxis(camRot[1], glm::vec3{ 0.0f, 1.0f, 0.0f });
		glm::mat4 rotMatY = glm::mat4_cast(rotQuat);
		rotQuat = glm::angleAxis(camRot[2], glm::vec3{ 0.0f, 0.0f, 1.0f });
		glm::mat4 rotMatZ = glm::mat4_cast(rotQuat);
		glm::vec4 lookVector = { 0, 0, 1, 0 };
		lookVector = rotMatX * rotMatY * rotMatZ * lookVector;

		// Camera physics

		std::vector<Triangle> tris = mesh.triangles;
		float closeT = cameraDist;
		float3 rayVec = normalize({ -cosf(playerRotY + glm::half_pi<float>()), 0.2f, -sinf(playerRotY + glm::half_pi<float>()) });

		for (int i = 0;i < ALLOC_MEM_TRIS_NUM;i++)
		{
			float3 p1{ tris[i].points[0], tris[i].points[1], tris[i].points[2] };
			float3 p2{ tris[i].points[3], tris[i].points[4], tris[i].points[5] };
			float3 p3{ tris[i].points[6], tris[i].points[7], tris[i].points[8] };
			float t;
			if (rayTriangleIntersect(p1, p2, p3, t, { playerPos[0], playerPos[1], playerPos[2] }, rayVec))
			{
				if (t < closeT)
				{
					closeT = t;
				}
			}
		}
		camPos[0] = playerPos[0] + max(closeT - 0.1f, 0.0f) * rayVec.x;
		camPos[1] = playerPos[1] + max(closeT - 0.1f, 0.0f) * rayVec.y;
		camPos[2] = playerPos[2] + max(closeT - 0.1f, 0.0f) * rayVec.z;

		// Player physics
		rayVec = { 0, -1, 0 };
		closeT = 0.2f;
		for (int i = 0;i < ALLOC_MEM_TRIS_NUM;i++)
		{
			float3 p1{ tris[i].points[0], tris[i].points[1], tris[i].points[2] };
			float3 p2{ tris[i].points[3], tris[i].points[4], tris[i].points[5] };
			float3 p3{ tris[i].points[6], tris[i].points[7], tris[i].points[8] };
			float t;
			if (rayTriangleIntersect(p1, p2, p3, t, { playerPos[0], playerPos[1], playerPos[2] }, rayVec))
			{
				if (t < closeT)
				{
					closeT = t;
					playerVec[0] = -tris[i].normal.x * 0.01f;
					playerVec[1] = -tris[i].normal.y * 0.015f;
					playerVec[2] = -tris[i].normal.z * 0.01f;
				}
			}
		}
		playerVec[1] -= 0.01f;
		playerVec[0] *= 0.97f;
		playerVec[2] *= 0.97f;
		playerPos[0] += playerVec[0];
		playerPos[1] += playerVec[1];
		playerPos[2] += playerVec[2];
		depth_UNDERCOVER += 0.5f;
		depth = (int)depth_UNDERCOVER;
		for (int y = 0;y < 4;y++)
		{
			Wrapper col = helper(y, camPos[0], camPos[1], camPos[2], sunPos[0], sunPos[1], sunPos[2], playerPos[0], playerPos[1], playerPos[2], camRot[0], camRot[1], &mesh.triangles[0], state, depth);
			for (int y2 = 0;y2 < RESOLUTION / 4;y2++)
			{
				for (int x = 0;x < RESOLUTION;x++)
				{

					cv::Vec3b& at = canvas.at<cv::Vec3b>(y2 + y * (RESOLUTION / 4), x);
					at.val[0] = col.r[(x + y2 * (RESOLUTION))];
					at.val[1] = col.g[(x + y2 * (RESOLUTION))];
					at.val[2] = col.b[(x + y2 * (RESOLUTION))];

				}
			}
		}
		mesh.triangles = oldTris;
		std::cout << (clock() - s) / CLOCKS_PER_SEC << std::endl;

		
		// Create a denoising filter
		if (depth_UNDERCOVER > 4.0f)
		{
			canvas.convertTo(canvas, CV_32FC3);
			filter.setImage("color", canvas.data, oidn::Format::Float3, RESOLUTION, RESOLUTION);
			filter.setImage("output", canvas.data, oidn::Format::Float3, RESOLUTION, RESOLUTION);
			filter.commit();
			filter.execute();
		}
		cv::imshow("Output", canvas);
		cv::setMouseCallback("Output", mouseCallback);
		cv::waitKey(1);
		if (GetKeyState('W') & 0x8000)
		{
			playerVec[2] += lookVector.z * 0.1;
			playerVec[0] += -lookVector.x * 0.1;
			playerVec[1] += lookVector.y * 0.1;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('S') & 0x8000)
		{
			playerVec[2] -= lookVector.z * 0.1;
			playerVec[0] -= -lookVector.x * 0.1;
			playerVec[1] -= lookVector.y * 0.1;
			depth_UNDERCOVER = 1.0f;
		}
		playerRotY = mouseDiff[0];
		camRot[1] = mouseDiff[0];
		if (GetKeyState('R') & 0x8000)
		{
			playerPos[1] -= 0.05f;
			depth_UNDERCOVER = 1.0f;

		}
		if (GetKeyState('T') & 0x8000)
		{
			playerPos[1] += 0.05f;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('F') & 0x8000)
		{
			sunPos[0] += 0.1f;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('G') & 0x8000)
		{
			sunPos[0] -= 0.1f;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('H') & 0x8000)
		{
			sunPos[1] += 0.1f;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('J') & 0x8000)
		{
			sunPos[1] -= 0.1f;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('K') & 0x8000)
		{
			sunPos[2] += 0.1f;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('L') & 0x8000)
		{
			sunPos[2] -= 0.1f;
			depth_UNDERCOVER = 1.0f;
		}
		if (GetKeyState('E') & 0x8000)
		{
			cameraDist -= 0.05f;
			depth_UNDERCOVER = 1.0f;

		}
		if (GetKeyState('Q') & 0x8000)
		{
			cameraDist += 0.05f;
			depth_UNDERCOVER = 1.0f;
		}
	}
	return;
}